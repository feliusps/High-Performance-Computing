#include "hip/hip_runtime.h"
//Felic feliu exercise 3 cuda using visual studio
/*C/Cuda program to accept a matrix of order M x N and find the movil average and average of each row and each column of a matrix*/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <memory>

#define SIZE 1024

//function 1 vectoravg_c column average
__global__ void vectoravg_c(int m, int n, float* data_d, float* avg_d)
{
	const int avg_index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if (avg_index >= n) {
		return;
	}

	float sum = 0;
	for (int i = 0; i < m; ++i)
	{
		sum = sum + data_d[i * n + avg_index];
	}
	avg_d[avg_index] = (sum / m);
}


//function 2 vectormov decrement loop
__global__ void vectormov(int m, int n, int w, float* data_d, float* mov_d)
{
	const int mov_index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	if (mov_index >= m * n) {
		return;
	}
	int index = mov_index;
	float sum = data_d[index];
	int count = 1;
	while (index % n != 0 && count < w - 1)
	{
		count++;
		index--;
		sum = sum + data_d[index];
	}
	mov_d[mov_index] = (sum / count);

}

int main()
{

	//Secuential part

	// initialize variables 

	int i, j, m, n;
	float* data = (float*)malloc(1000 * 1000 * sizeof(float));
	float* mov = (float*)malloc(1000 * 1000 * sizeof(float));
	float* avg_c = (float*)malloc(1000 * sizeof(float));
	memset(data, 0, 1000 * 1000 * sizeof(float));
	memset(mov, 0, 1000 * 1000 * sizeof(float));
	memset(avg_c, 0, 1000 * sizeof(float));


	printf("Enter the order of the matrix\n");
	scanf_s("%d %d", &m, &n);




	FILE* myFile;
	fopen_s(&myFile, "ooi_temp7m.txt", "r");
	if (myFile == NULL) {
		printf("cannot open the file\n");
		return 1;
	}
	for (i = 0; i < m; ++i)
	{
		for (j = 0; j < n; ++j)
		{
			int index = i * n + j;
			if (fscanf(myFile, "%f", &data[index]) == EOF) {
				fseek(myFile, 0, SEEK_SET);
			}
		}
	}

	// CUDA part 

	//initialized variables
	float* array_d;
	float* mov_d, * avg_d;

	int size = m * n * sizeof(float);
	int size1 = m * n * sizeof(float);
	int size2 = n * sizeof(float);

	// Allocate memory on device
	hipMalloc(&array_d, size);
	hipMalloc(&mov_d, size1);
	hipMalloc(&avg_d, size2);

	// initilized matrix on device
	hipMemcpy(array_d, data, size, hipMemcpyHostToDevice);
	hipMemset(mov_d, 0, size1);
	hipMemset(avg_d, 0, size2);

	// call kernel function 1 vectoravg increment loop
	int bloques = (n + 128 - 1) / 128;
	int y_bloques;
	int x_bloques;

	if (bloques > 128) {
		y_bloques = (bloques + 128 - 1) / 128;
		x_bloques = 128;
	}
	else {
		y_bloques = 1;
		x_bloques = 128;

	}
	dim3 tamGrid1(x_bloques, y_bloques); //grid dimensi�n
	dim3 tamBlock1(128, 1, 1); //glock dimensi�n

							  // launch the device computation threads!
	vectoravg_c << <tamGrid1, tamBlock1 >> > (m, n, array_d, avg_d);
	hipDeviceSynchronize();

	// call kernel function 2 vectormov decrement loop
	bloques = (m * n + 128 - 1) / 128;
	if (bloques > 128) {
		y_bloques = (bloques + 128 - 1) / 128;
		x_bloques = 128;
	}
	else {
		y_bloques = 1;
		x_bloques = 128;

	}
	dim3 tamGrid2(x_bloques, y_bloques); //grid dimension
	dim3 tamBlock2(128, 1, 1); //block dimension

	// Launch the device computation threads!
	vectormov << <tamGrid2, tamBlock2 >> > (m, n, 9, array_d, mov_d);

	hipDeviceSynchronize();

	// bring results;
	hipMemcpy(avg_c, avg_d, size2, hipMemcpyDeviceToHost);
	hipMemcpy(mov, mov_d, size1, hipMemcpyDeviceToHost);

	// free memory in  device
	hipFree(avg_d);
	hipFree(mov_d);
	hipFree(array_d);



	//	print mov matrix
	FILE* file;
	fopen_s(&file, "mov_avg_cyberinfrastructure.csv", "w"); //if does not  work, try file=fopen("vectors.csv", "w");


	fprintf(file, "MOV[M,N] : \n");
	for (i = 0; i < m; ++i)
	{
		for (j = 0; j < n; ++j)
		{
			int index = i * n + j;
			if (j != n - 1)
				fprintf(file, "%.5f,", mov[index]);
			else
				fprintf(file, "%.5f", mov[index]);
		}

		fprintf(file, "\n");
	}

	//	print mov matrix
	fprintf(file, "AVG[N] : \n");
	for (j = 0; j < n; ++j)
	{
		if (j != n - 1)
			fprintf(file, "%.5f,", avg_c[j]);
		else
			fprintf(file, "%.5f", avg_c[j]);
	}

	fclose(file);



	return 0;
}

